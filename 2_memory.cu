#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 100

// __global__ = runs on GPU, called from CPU
// __device__ = runs on GPU, called from GPU
// __host__ = runs on CPU, called from CPU (default)

__global__ void fun(int *a){
    // Each thread adds 10 to one array element
    // threadIdx.x ranges from 0 to N-1 (since we launch N threads)
    // 
    // Memory access pattern: COALESCED
    // Thread 0 accesses a[0], Thread 1 → a[1], etc.
    // Consecutive threads → consecutive memory = fast!
    a[threadIdx.x] += 10;
}

int main(){
    // Step 1: Allocate CPU (host) memory
    int a[N];
    int *d_a;  // 'd_' prefix = device pointer (convention)
    
    for(int i=0;i<N;i++) a[i] = i;  // a = [0,1,2,...,99]

    // Step 2: Allocate GPU (device) memory
    // cudaMalloc allocates GLOBAL memory on GPU
    // Global memory: ~400-800 cycle latency, 4-24 GB size
    hipMalloc((void**)&d_a, N*sizeof(int));
    
    // Step 3: Copy data CPU → GPU (via PCIe bus)
    // This is SLOW (~10 GB/s vs ~900 GB/s GPU memory bandwidth)
    // Minimize these transfers!
    hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);

    // Step 4: Launch kernel
    // <<<1, N>>> = 1 block, N threads
    // All N threads run in parallel (grouped into ⌈N/32⌉ warps)
    fun<<<1, N>>>(d_a); 
    
    // Step 5: Copy result GPU → CPU
    hipMemcpy(a, d_a, N*sizeof(int), hipMemcpyDeviceToHost);
    
    // Step 6: Free GPU memory (prevents memory leak)
    hipFree(d_a);

    // Print result: should be [10,11,12,...,109]
    for(int i=0;i<N;i++) printf("%d ", a[i]);
    printf("\n");
    
    // Memory Hierarchy Used:
    // 1. Registers: threadIdx.x stored in register
    // 2. Global Memory: array 'a' accessed (slowest)
    // 3. L2 Cache: automatic caching helps repeated access
    //
    // Optimization opportunity:
    // → Each thread only accesses memory once (good!)
    // → Could use shared memory if threads needed to communicate
    
    return 0;
}

// Performance Notes:
// - cudaMemcpy time >> kernel execution time for small N
// - Rule of thumb: Transfer once, compute much
// - For N=100: kernel ~1 microsecond, memcpy ~10 microseconds